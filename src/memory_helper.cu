#include "memory_helper.h"
using namespace std;

void* MemoryMonitor::cpuMalloc(int size){
	cpuMemory += size;
	void* p = malloc(size);
	cpuPoint[p] = 1.0f * size;
 	//if(size >= 1024 * 1024){
 	//	printf("******************************* cpu malloc memory %fMb\n", 1.0 * size / 1024 / 1024);
 	//}
	return p;
}

void MemoryMonitor::freeCpuMemory(void* ptr)
{
	if(cpuPoint.find(ptr) != cpuPoint.end()){
 		//if(cpuPoint[ptr] >= 1024 * 1024){
 		//	printf("+++++++++++++++++++++++++++++++ free cpu memory %fMb\n", cpuPoint[ptr] / 1024 / 1024);
 		//}
		cpuMemory -= cpuPoint[ptr];
		free(ptr);
		cpuPoint.erase(ptr);
	}
}

hipError_t MemoryMonitor::gpuMalloc(void** devPtr, int size){
	gpuMemory += size;
	hipError_t error = hipMalloc(devPtr, size);
	gpuPoint[*devPtr] = (float)size;
 	//if(size >= 1024 * 1024){
 	//	printf("******************************* gpu malloc memory %fMb\n", 1.0 * size / 1024 / 1024);
 	//}
	//printf("******************************* using gpu memory %fMb\n", gpuMemory / 1024 / 1024);
	return error;
}

void MemoryMonitor::freeGpuMemory(void* ptr){
	if(gpuPoint.find(ptr) != gpuPoint.end()){
 		//if(gpuPoint[ptr] >= 1024 * 1024){
 		//	printf("+++++++++++++++++++++++++++++++ free gpu memory %fMb\n", gpuPoint[ptr] / 1024 / 1024);
 		//}
		gpuMemory -= gpuPoint[ptr];
		hipFree(ptr);
		gpuPoint.erase(ptr);
		//printf("******************************* using gpu memory %fMb\n", gpuMemory / 1024 / 1024);
	}
}


float MemoryMonitor::getCpuMemory() const{
	return cpuMemory;
}

float MemoryMonitor::getGpuMemory() const{
	return gpuMemory;
}
