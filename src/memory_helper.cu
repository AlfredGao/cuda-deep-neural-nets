#include "memory_helper.h"
using namespace std;

void* MemoryMonitor::cpuMalloc(int size){
	cpuMemory += size;
	void* p = malloc(size);
	cpuPoint[p] = 1.0f * size;
 	//if(size >= 1024 * 1024){
 	//	printf("******************************* cpu malloc memory %fMb\n", 1.0 * size / 1024 / 1024);
 	//}
	return p;
}

void MemoryMonitor::freeCpuMemory(void* ptr)
{
	if(cpuPoint.find(ptr) != cpuPoint.end()){
 		//if(cpuPoint[ptr] >= 1024 * 1024){
 		//	printf("+++++++++++++++++++++++++++++++ free cpu memory %fMb\n", cpuPoint[ptr] / 1024 / 1024);
 		//}
		cpuMemory -= cpuPoint[ptr];
		free(ptr);
		cpuPoint.erase(ptr);
	}
}

hipError_t MemoryMonitor::gpuMalloc(void** devPtr, int size){
	//gpuMemory += size;
	//hipError_t error = hipMalloc(devPtr, size);
	//gpuPoint[*devPtr] = (float)size;

	const size_t Mb = 1<<20; // Assuming a 1Mb page size here
	size_t available, total;
	hipMemGetInfo(&available, &total);
	size_t nwords = total / sizeof(float);
	size_t words_per_Mb = Mb / sizeof(float);
	hipError_t error;
	while(1){
		error = hipMalloc(devPtr,  size);
		if(hipSuccess != error){
		 	printf("******************************* gpu malloc memory %fMb\n", 1.0 * size / 1024 / 1024);
		}
		if(hipErrorOutOfMemory != error) break;
	    nwords -= words_per_Mb;
	    if( nwords  < words_per_Mb){
	        // signal no free memory
	    	return error;
	    }
	}
	gpuMemory += size;
	gpuPoint[*devPtr] = (float)size;
	return error;
 	//if(size >= 1024 * 1024){
 	//	printf("******************************* gpu malloc memory %fMb\n", 1.0 * size / 1024 / 1024);
 	//}
}

void MemoryMonitor::freeGpuMemory(void* ptr){
	if(gpuPoint.find(ptr) != gpuPoint.end()){
 		//if(gpuPoint[ptr] >= 1024 * 1024){
 		//	printf("+++++++++++++++++++++++++++++++ free gpu memory %fMb\n", gpuPoint[ptr] / 1024 / 1024);
 		//}
		gpuMemory -= gpuPoint[ptr];
		hipFree(ptr);
		gpuPoint.erase(ptr);
	}
}


float MemoryMonitor::getCpuMemory() const{
	return cpuMemory;
}

float MemoryMonitor::getGpuMemory() const{
	return gpuMemory;
}
