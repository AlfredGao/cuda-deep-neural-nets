#include "hip/hip_runtime.h"
#include "cu_matrix_maths.h"

__global__ void cu_plus(float *A, const float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fadd_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_plus(const float *A, const float *B, float *C, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		C[tid] = __fadd_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_plus(float *A, const float b, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fadd_rd(A[tid], b);
		tid += stride;
	}
}

__global__ void cu_plus(const float *A, float *B, const float c, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		B[tid] = __fadd_rd(A[tid], c);
		tid += stride;
	}
}

__global__ void cu_minus(float *A, const float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fsub_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_minus(const float *A, const float *B, float *C, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		C[tid] = __fsub_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_minus(float *A, const float b, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fsub_rd(A[tid], b);
		tid += stride;
	}
}

__global__ void cu_minus(const float *A, float *B, const float c, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		B[tid] = __fsub_rd(A[tid], c);
		tid += stride;
	}
}

__global__ void cu_square(const float *A, float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		B[tid] = __fmul_rd(A[tid], A[tid]);
		tid += stride;
	}
}

__global__ void cu_sqrt(const float *A, float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		B[tid] = sqrtf(A[tid]);
		tid += stride;
	}
}

__global__ void cu_elementWiseMultiply(float *A, const float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fmul_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_elementWiseMultiply(float *A, float B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fmul_rd(A[tid], B);
		tid += stride;
	}
}

__global__ void cu_elementWiseMultiply(const float *A, const float *B, float *C, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		C[tid] = __fmul_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_elementWiseMultiply(const float *A, const float B, float *C, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		C[tid] = __fmul_rd(A[tid], B);
		tid += stride;
	}
}

__global__ void cu_setAll(float* A, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = val;
		tid += stride;
	}
}

__global__ void cu_exp(const float* src, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		dst[tid] = __expf(src[tid]);
		tid += stride;
	}
}

__global__ void cu_log(const float* src, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		dst[tid] = __logf(src[tid]);
		tid += stride;
	}
}

__global__ void cu_pow(const float* src, float* dst, const float power, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		dst[tid] = powf(src[tid], power);
		tid += stride;
	}
}

__global__ void cu_divide(float *A, float B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fdividef(A[tid], B);
		tid += stride;
	}
}

__global__ void cu_divide(const float* src, float* dst, const float denominator, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(0 == denominator) dst[tid] = 0.0;
		else dst[tid] = __fdividef(src[tid], denominator);
		tid += stride;
	}
}

__global__ void cu_divide(const float numerator, const float* src, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(0 == src[tid]) dst[tid] = 0.0;
		else dst[tid] = __fdividef(numerator, src[tid]);
		tid += stride;
	}
}

__global__ void cu_divide(const float* numerator, const float* denominator, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(0 == denominator[tid]) dst[tid] = 0.0;
		else dst[tid] = __fdividef(numerator[tid], denominator[tid]);
		tid += stride;
	}
}

__global__ void cu_sum(const float* src, float* sum, const int n){
	extern __shared__ float sdata[];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// load input into __shared__ memory
	float x = 0;
	if(tid < n){
		x = src[tid];
	}
	sdata[threadIdx.x] = x;
	__syncthreads();
	// contiguous range pattern
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1){
		if(threadIdx.x < offset){
			// add a partial sum upstream to our own
			sdata[threadIdx.x] += sdata[threadIdx.x + offset];
		}
	    // wait until all threads in the block have
	    // updated their partial sums
		__syncthreads();
	}
	// thread 0 writes the final result
	if(threadIdx.x == 0){
		sum[blockIdx.x] = sdata[0];
	}
	__syncthreads();
}

__global__ void cu_minMaxLoc(const float* src, float* minValue, float* maxValue, int* minLoc, int* maxLoc, const int n){
	__shared__ float minValCache[threadsPerBlock];
	__shared__ float maxValCache[threadsPerBlock];
	__shared__ int minLocCache[threadsPerBlock];
	__shared__ int maxLocCache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int stride = blockDim.x * gridDim.x;
	float val = src[0];
	int loc = 0;
	if(tid < n){
		val = src[tid];
		loc = tid;
	}
	maxValCache[threadIdx.x] = val;
	minValCache[threadIdx.x] = val;
	maxLocCache[threadIdx.x] = loc;
	minLocCache[threadIdx.x] = loc;
	__syncthreads();
	// contiguous range pattern
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1){
		if(threadIdx.x < offset){
			// add a partial sum upstream to our own
			if(maxValCache[threadIdx.x] >= maxValCache[threadIdx.x + offset]){
				;
			}else{
				maxValCache[threadIdx.x] = maxValCache[threadIdx.x + offset];
				maxLocCache[threadIdx.x] = maxLocCache[threadIdx.x + offset];
			}
			if(minValCache[threadIdx.x] <= minValCache[threadIdx.x + offset]){
				;
			}else{
				minValCache[threadIdx.x] = minValCache[threadIdx.x + offset];
				minLocCache[threadIdx.x] = minLocCache[threadIdx.x + offset];
			}
		}
	    // wait until all threads in the block have
	    // updated their partial sums
		__syncthreads();
	}
	// thread 0 writes the final result
	if(threadIdx.x == 0){
		minValue[blockIdx.x] = minValCache[0];
		maxValue[blockIdx.x] = maxValCache[0];
		minLoc[blockIdx.x] = minLocCache[0];
		maxLoc[blockIdx.x] = maxLocCache[0];
	}
}


__global__ void cu_greaterThan(const float* src, float* dst, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] > val) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_greaterThanOrEqualTo(const float* src, float* dst, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] >= val) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_lessThan(const float* src, float* dst, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] < val) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_lessThanOrEqualTo(const float* src, float* dst, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] <= val) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_equalTo(const float* src, float* dst, const float val, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] == val) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_fliplr(const float* src, float* dst, const int rows, const int cols, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int c = tid % cols;
		int r = tid / cols;
		dst[tid] = src[(cols - c - 1) + r * cols];
		tid += stride;
	}
}

__global__ void cu_padding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int pad = (cols2 - cols1) / 2;
		int c1 = tid % cols1;
		int r1 = tid / cols1;
		int r2 = r1 + pad;
		int c2 = c1 + pad;
		dst[r2 * cols2 + c2] = src[tid];
		tid += stride;
	}
}

__global__ void cu_depadding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int pad = (cols1 - cols2) / 2;
		int c2 = tid % cols2;
		int r2 = tid / cols2;
		int r1 = r2 + pad;
		int c1 = c2 + pad;
		dst[tid] = src[r1 * cols1 + c1];
		tid += stride;
	}
}

__global__ void cu_repmat(const float *a, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int c2 = tid % colsdst;
		int r2 = tid / colsdst;
		int ra = r2 % rowsa;
		int ca = c2 % colsa;
		dst[tid] = a[ra * colsa + ca];
		tid += stride;
	}
}

__global__ void cu_kron(const float *a, const float* b, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int colsb = colsdst / colsa;
	int rowsb = rowsdst / rowsa;
	while(tid < n){
		int c2 = tid % colsdst;
		int r2 = tid / colsdst;
		int rb = r2 % rowsb;
		int cb = c2 % colsb;
		int ra = r2 / rowsb;
		int ca = c2 / colsb;
		dst[tid] = a[ra * colsa + ca] * b[rb * colsb + cb];
		tid += stride;
	}
}

__global__ void cu_downSample(const float *src, float* dst, const int y_stride, const int x_stride, const int colssrc, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int colsdst = colssrc / x_stride;
	if(colssrc % x_stride > 0) ++colsdst;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = rdst * y_stride;
		int csrc = cdst * x_stride;
		dst[tid] = src[rsrc * colssrc + csrc];
		tid += stride;
	}
}

__global__ void cu_interpolation(const float* src, float* dst, const int colssrc, const int colsdst, const int _stride, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int csrc = tid % colssrc;
		int rsrc = tid / colssrc;
		int rdst = rsrc * _stride;
		int cdst = csrc * _stride;
		dst[rdst * colsdst + cdst] = src[tid];
		tid += stride;
	}
}

__global__ void cu_getRange(const float *src, float* dst, const int xstart, const int xend, const int ystart, const int yend, const int colssrc, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int colsdst = xend - xstart + 1;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = rdst + ystart;
		int csrc = cdst + xstart;
		dst[tid] = src[rsrc * colssrc + csrc];
		tid += stride;
	}
}

__global__ void cu_copyMakeBorder(const float *src, float* dst, const int rowssrc, const int colssrc, const int up, const int down, const int left, const int right, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int colsdst = colssrc + left + right;
	while(tid < n){
		int csrc = tid % colssrc;
		int rsrc = tid / colssrc;
		int rdst = up + rsrc;
		int cdst = left + csrc;
		dst[rdst * colsdst + cdst] = src[tid];
		tid += stride;
	}
}

__global__ void cu_pooling_max(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int stridex, const int stridey, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = rdst * stridey;
		int csrc = cdst * stridex;
		int xend = (csrc + stridex - 1) > (colssrc - 1) ? (colssrc - 1) : (csrc + stridex - 1);
		int yend = (rsrc + stridey - 1) > (rowssrc - 1) ? (rowssrc - 1) : (rsrc + stridey - 1);
		loc[tid] = (float)(rsrc * colssrc + csrc);
		for(int i = rsrc; i <= yend; ++i){
			for(int j = csrc; j <= xend; ++j){
				if(src[i * colssrc + j] > dst[tid]){
					dst[tid] = src[i * colssrc + j];
					loc[tid] = (float)(i * colssrc + j);
				}
			}
		}
		tid += stride;
	}
}

__global__ void cu_pooling_mean(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int stridex, const int stridey, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = rdst * stridey;
		int csrc = cdst * stridex;
		int xend = (csrc + stridex - 1) > (colssrc - 1) ? (colssrc - 1) : (csrc + stridex - 1);
		int yend = (rsrc + stridey - 1) > (rowssrc - 1) ? (rowssrc - 1) : (rsrc + stridey - 1);
		loc[tid] = (float)(rsrc * colssrc + csrc);
		for(int i = rsrc; i <= yend; ++i){
			for(int j = csrc; j <= xend; ++j){
				dst[tid] += __fdividef(src[i * colssrc + j], __fmul_rd(yend - rsrc + 1, xend - csrc + 1));	
			}
		}
		tid += stride;
	}
}

__global__ void cu_pooling_overlap_max(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int sizex, const int sizey, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = rdst;
		int csrc = cdst;
		int xend = (csrc + sizex - 1);
		int yend = (rsrc + sizey - 1);
		loc[tid] = (float)(rsrc * colssrc + csrc);
		for(int i = rsrc; i <= yend; ++i){
			for(int j = csrc; j <= xend; ++j){
				if(src[i * colssrc + j] > dst[tid]){
					dst[tid] = src[i * colssrc + j];
					loc[tid] = (float)(i * colssrc + j);
				}
			}
		}
		tid += stride;
	}
}

__global__ void cu_unpooling(const float* src, const float* loc, float* dst, const int colsdst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int cdst = (int)(loc[tid]) % colsdst;
		int rdst = (int)(loc[tid]) / colsdst;
		dst[rdst * colsdst + cdst] = src[tid];
		tid += stride;
	}
}

__global__ void cu_multiply(const float* A, const float* B, float * C,
                                    int rowsa, int colsa,
                                    int rowsb, int colsb,
                                    int rowsc, int colsc){
    __shared__ float sA[32][32];   // Tile size of 32x32
    __shared__ float sB[32][32];
    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;
    for (int k = 0; k < (((colsa - 1)/ 32) + 1); k++){
        if ( (Row < rowsa) && (threadIdx.x + (k*32)) < colsa){
            sA[threadIdx.y][threadIdx.x] = A[(Row*colsa) + threadIdx.x + (k*32)];
        }
        else{
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();
        if ( Col < colsb && (threadIdx.y + k*32) < rowsb){
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*32)*colsb + Col];
        }
        else{
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < 32; ++j){
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
        __syncthreads();
    }
    if (Row < rowsc && Col < colsc){
        C[Row*colsc + Col] = Cvalue;
    }
}

__global__ void cu_transpose(const float* src, float* dst, int colssrc, int colsdst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		int cdst = tid % colsdst;
		int rdst = tid / colsdst;
		int rsrc = cdst;
		int csrc = rdst;
		dst[tid] = src[rsrc * colssrc + csrc];
		tid += stride;
	}
}

__global__ void cu_sigmoid(const float* src, float* dst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float tmp = __fmul_rd(src[tid], -1.0);
		tmp = __expf(tmp);
		tmp = __fadd_rd(tmp, 1.0);
		dst[tid] = __fdividef(1.0, tmp);
		tid += stride;
	}
}

__global__ void cu_dsigmoid(const float* src, float* dst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float tmp = __expf(src[tid]);
		float tmp2 = __fadd_rd(tmp, 1.0);
		tmp2 = __fmul_rd(tmp2, tmp2);
		dst[tid] = fdividef(tmp, tmp2);
		tid += stride;
	}
}

__global__ void cu_dsigmoid_a(const float* src, float* dst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float tmp = __fsub_rd(1.0, src[tid]);
		dst[tid] = __fmul_rd(tmp, src[tid]);
		tid += stride;
	}
}

__global__ void cu_relu(const float* src, float* dst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] > 0.0) dst[tid] = src[tid];
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_drelu(const float* src, float* dst, int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		if(src[tid] > 0.0) dst[tid] = 1.0;
		else dst[tid] = 0.0;
		tid += stride;
	}
}

__global__ void cu_leaky_relu(const float* src, float* dst, int n){
	const float leaky_relu_alpha = 100.0;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float p = 0.0;
		float n = 0.0;
		if(src[tid] > 0.0) p = src[tid];
		if(src[tid] < 0.0) n = src[tid];
		n = fdividef(n, leaky_relu_alpha);
		dst[tid] = __fadd_rd(p, n);
		tid += stride;
	}
}

__global__ void cu_dleaky_relu(const float* src, float* dst, int n){
	const float leaky_relu_alpha = 100.0;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float p = 0.0;
		float n = 0.0;
		if(src[tid] > 0.0) p = 1;
		if(src[tid] < 0.0) n = 1;
		n = fdividef(n, leaky_relu_alpha);
		dst[tid] = __fadd_rd(p, n);
		tid += stride;
	}
}

__global__ void cu_tanh(const float* src, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		dst[tid] = tanhf(src[tid]);
		tid += stride;
	}
}

__global__ void cu_dtanh(const float* src, float* dst, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		float tmp = __fmul_rd(src[tid], src[tid]);
		dst[tid] = __fsub_rd(1.0, tmp);
		tid += stride;
	}
}










