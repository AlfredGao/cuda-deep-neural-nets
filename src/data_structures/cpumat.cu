#include "data_structure.h"

using namespace std;

cpuMat::cpuMat(){
	rows = 0;
	cols = 0;
	channels = 0;
	Data = NULL;
}

cpuMat::cpuMat(const cpuMat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	Data = NULL;
	mallocMat();
	memcpy(Data, m.Data, getLength() * sizeof(float));
}

cpuMat::cpuMat(const Mat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	Data = NULL;
	mallocMat();
	memcpy(Data, m.hostData, getLength() * sizeof(float));
}

cpuMat::cpuMat(int height, int width, int nchannels){
	cols = width;
	rows = height;
	channels = nchannels;
	Data = NULL;
	mallocMat();
	zeros();
}
cpuMat::~cpuMat(){
	if(NULL != Data)
		MemoryMonitor::instance()->freeCpuMemory(Data);
}

void cpuMat::release(){
	if(NULL != Data)
		MemoryMonitor::instance()->freeCpuMemory(Data);
	rows = 0;
	cols = 0;
	channels = 0;
	Data = NULL;
}

cpuMat& cpuMat::operator=(const cpuMat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	if(NULL != Data){
		MemoryMonitor::instance()->freeCpuMemory(Data);
		Data = NULL;
	}
	mallocMat();
	memcpy(Data, m.Data, getLength() * sizeof(float));
    return *this;
}

cpuMat& cpuMat::operator<<=(cpuMat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	if(NULL != Data){
		MemoryMonitor::instance()->freeCpuMemory(Data);
		Data = NULL;
	}
	mallocMat();
	memcpy(Data, m.Data, getLength() * sizeof(float));
	m.release();
    return *this;
}

void cpuMat::setSize(int r, int c, int ch){
	rows = r;
	cols = c;
	channels = ch;
	if(NULL != Data){
		MemoryMonitor::instance()->freeCpuMemory(Data);
		Data = NULL;
	}
	mallocMat();
	zeros();
}

void cpuMat::zeros(){
	setAll(0.0);
}

void cpuMat::ones(){
	setAll(1.0);
}

void cpuMat::randu(){
	if(NULL == Data) mallocMat();
	for(int i = 0; i < rows; ++i){
		for(int j = 0; j < cols; ++j){
			for(int ch = 0; ch < channels; ++ch){
				set(i, j, ch, 2.0 * ((float) rand() / (RAND_MAX)) - 1.0);
			}
		}
	}
}

void cpuMat::set(int pos_y, int pos_x, int pos_channel, float val){
	if(NULL == Data) {zeros();}
	if(pos_x >= cols || pos_y >= rows || pos_channel >= channels){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	Data[RC2IDX(pos_y, pos_x, cols) + pos_channel * (rows * cols)] = val;
}

void cpuMat::set(int pos_y, int pos_x, const vector3f& val){
	if(NULL == Data ) {zeros();}
	if(pos_x >= cols || pos_y >= rows){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	for(int i = 0; i < channels; ++i){
		set(pos_y, pos_x, i, val.get(i));
	}
}

void cpuMat::set(int pos, const vector3f& val){
	if(NULL == Data ) {zeros();}
	if(pos >= cols * rows){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	for(int i = 0; i < channels; ++i){
		Data[pos + i * (rows * cols)] = val.get(i);
	}
}

void cpuMat::set(int pos, int pos_channel, float val){
	if(NULL == Data ) {zeros();}
	if(pos >= cols * rows){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	Data[pos + pos_channel * (rows * cols)] = val;
}

void cpuMat::setAll(float val){
	if(NULL == Data) {mallocMat();}
	int len = getLength();
	for(int i = 0; i < len; ++i){
		Data[i] = val;
	}
}

void cpuMat::setAll(const vector3f &v){
	if(NULL == Data) {mallocMat();}
	int len = rows * cols;
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < len; ++i){
			Data[len * ch + i] = v.get(ch);
		}
	}
}

float cpuMat::get(int pos_y, int pos_x, int pos_channel) const{
	if(NULL == Data ||
	   pos_x >= cols || pos_y >= rows || pos_channel >= channels){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	return Data[RC2IDX(pos_y, pos_x, cols) + pos_channel * (rows * cols)];
}

vector3f cpuMat::get(int pos_y, int pos_x) const{
	if(NULL == Data ||
	   pos_x >= cols || pos_y >= rows || channels < 3){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	vector3f res;
	for(int i = 0; i < 3; ++i){
		res.set(i, Data[RC2IDX(pos_y, pos_x, cols) + i * (rows * cols)]);
	}
	return res;
}

int cpuMat::getLength() const{
	return rows * cols * channels;
}

void cpuMat::copyTo(cpuMat &m) const{
	m.rows = rows;
	m.cols = cols;
	m.channels = channels;
	if(NULL != m.Data){
		MemoryMonitor::instance()->freeCpuMemory(m.Data);
		m.Data = NULL;
	}
	m.mallocMat();
	memcpy(m.Data, Data, getLength() * sizeof(float));
}

void cpuMat::copyTo(Mat &m) const{
	m.rows = rows;
	m.cols = cols;
	m.channels = channels;
	if(NULL != m.hostData){
		MemoryMonitor::instance()->freeCpuMemory(m.hostData);
		m.hostData = NULL;
	}
	if(NULL != m.devData){
		MemoryMonitor::instance()->freeGpuMemory(m.devData);
		m.devData = NULL;
	}
	m.mallocHost();
	m.mallocDevice();
	memcpy(m.hostData, Data, getLength() * sizeof(float));
	hipMemcpy(m.devData, Data, getLength() * sizeof(float), hipMemcpyHostToDevice);
}

void cpuMat::moveTo(cpuMat &m){
	m.rows = rows;
	m.cols = cols;
	m.channels = channels;
	if(NULL != m.Data){
		MemoryMonitor::instance()->freeCpuMemory(m.Data);
		m.Data = NULL;
	}
	m.mallocMat();
	memcpy(m.Data, Data, getLength() * sizeof(float));
	release();
}

void cpuMat::moveTo(Mat &m){
	m.rows = rows;
	m.cols = cols;
	m.channels = channels;
	if(NULL != m.hostData){
		MemoryMonitor::instance()->freeCpuMemory(m.hostData);
		m.hostData = NULL;
	}
	if(NULL != m.devData){
		MemoryMonitor::instance()->freeGpuMemory(m.devData);
		m.devData = NULL;
	}
	m.mallocHost();
	m.mallocDevice();
	memcpy(m.hostData, Data, getLength() * sizeof(float));
	hipMemcpy(m.devData, Data, getLength() * sizeof(float), hipMemcpyHostToDevice);
	release();
}

// only changes devData (on GPU)
cpuMat cpuMat::operator+(const cpuMat &m) const{
	if(NULL == Data  || NULL == m.Data|| getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(m);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = tmp.Data[i] + Data[i];
	}
	return tmp;
}

cpuMat cpuMat::operator+(float val) const{
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(rows, cols, channels);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] + val;
	}
	return tmp;
}

cpuMat cpuMat::operator+(const vector3f &v) const{
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	cpuMat tmp(rows, cols, channels);
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			tmp.Data[i + n * ch] = Data[i + n * ch] + v.get(ch);
		}
	}
	return tmp;
}

cpuMat& cpuMat::operator+=(const cpuMat &m){
	if(NULL == Data  || NULL == m.Data|| getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	for(int i = 0; i < n; ++i){
		Data[i] += m.Data[i];
	}
	return *this;
}

cpuMat& cpuMat::operator+=(float val) {
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	for(int i = 0; i < n; ++i){
		Data[i] += val;
	}
	return *this;
}

cpuMat& cpuMat::operator+=(const vector3f &v){
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			Data[i + n * ch] += v.get(ch);
		}
	}
	return *this;
}

cpuMat cpuMat::operator-(const cpuMat &m) const{

	if(NULL == Data  || NULL == m.Data || getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(m);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] - tmp.Data[i];
	}
	return tmp;
}

cpuMat cpuMat::operator-(float val) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(rows, cols, channels);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] - val;
	}
	return tmp;
}

cpuMat cpuMat::operator-(const vector3f& v) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	cpuMat tmp(rows, cols, channels);
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			tmp.Data[i + n * ch] = Data[i + n * ch] - v.get(ch);
		}
	}
	return tmp;
}

cpuMat& cpuMat::operator-=(const cpuMat &m){
	if(NULL == Data  || NULL == m.Data|| getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	for(int i = 0; i < n; ++i){
		Data[i] -= m.Data[i];
	}
	return *this;
}

cpuMat& cpuMat::operator-=(float val) {
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	for(int i = 0; i < n; ++i){
		Data[i] -= val;
	}
	return *this;
}

cpuMat& cpuMat::operator-=(const vector3f &v){
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			Data[i + n * ch] -= v.get(ch);
		}
	}
	return *this;
}
/*
cpuMat cpuMat::operator*(const cpuMat &m){
	if(NULL == Data  ||
	   NULL == m.Data ||
	   cols != m.rows || channels != m.channels){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
}
*/

cpuMat cpuMat::operator*(float val) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(rows, cols, channels);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] * val;
	}
	return tmp;
}

cpuMat cpuMat::operator*(const vector3f &v) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	cpuMat tmp(rows, cols, channels);
	for(int ch = 0; ch < 3; ++ch){
		for(int i = 0; i < channels; ++i){
			tmp.Data[i + n * ch] = Data[i + n * ch] * v.get(ch);
		}
	}
	return tmp;
}

cpuMat& cpuMat::operator*=(float val) {
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	for(int i = 0; i < n; ++i){
		Data[i] *= val;
	}
	return *this;
}

cpuMat& cpuMat::operator*=(const vector3f &v){
	if(NULL == Data){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			Data[i + n * ch] *= v.get(ch);
		}
	}
	return *this;
}

cpuMat cpuMat::mul(const cpuMat &m) const{
	if(NULL == Data  || NULL == m.Data || getLength()!= m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(m);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] * tmp.Data[i];
	}
	return tmp;
}

cpuMat cpuMat::mul(float val) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	cpuMat tmp(rows, cols, channels);
	for(int i = 0; i < n; ++i){
		tmp.Data[i] = Data[i] * val;
	}
	return tmp;
}

cpuMat cpuMat::mul(const vector3f &v) const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = rows * cols;
	cpuMat tmp(rows, cols, channels);
	for(int ch = 0; ch < channels; ++ch){
		for(int i = 0; i < n; ++i){
			tmp.Data[i + n * ch] = Data[i + n * ch] * v.get(ch);
		}
	}
	return tmp;
}


cpuMat cpuMat::t() const{
	if(NULL == Data ){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	cpuMat tmp(cols, rows, channels);
	for(int i = 0; i < tmp.rows; ++i){
		for(int j = 0; j < tmp.cols; ++j){
			tmp.set(i, j, get(j, i));
		}
	}
	return tmp;
}

// memory
void cpuMat::mallocMat(){
	if(NULL == Data){
		// malloc host data
		Data = (float*)MemoryMonitor::instance()->cpuMalloc(cols * rows * channels * sizeof(float));
		if(NULL == Data) {
			std::cout<<"host memory allocation failed..."<<std::endl;
			exit(0);
		}
		memset(Data, 0, cols * rows * channels * sizeof(float));
	}
}

void cpuMat::print(const std::string &str) const{
	std::cout<<str<<std::endl;
	if(NULL == Data ){
		std::cout<<"invalid cpuMatrix..."<<std::endl;
		exit(0);
	}
	int counter = 0;
	std::cout<<"cpuMatrix with "<<channels<<" channels, "<<rows<<" rows, "<<cols<<"columns."<<std::endl;
	for(int i = 0; i < channels; ++i){
		std::cout<<"Channel "<<i<<" : "<<std::endl;
		for(int j = 0; j < rows; ++j){
			for(int k = 0; k < cols; ++k){
				std::cout<<Data[counter]<<" ";
				++ counter;
			}
			std::cout<<std::endl;
		}
	}
}
