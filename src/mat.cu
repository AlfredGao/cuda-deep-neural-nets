#include "data_structure.h"

using namespace std;

Mat::Mat(){
	rows = 0;
	cols = 0;
	channels = 0;
	hostData = NULL;
	devData = NULL;
}
Mat::Mat(const Mat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	hostData = NULL;
	devData = NULL;
	mallocHost();
	mallocDevice();
	memcpy(hostData, m.hostData, getLength() * sizeof(float));
	hipMemcpy(devData, m.devData, getLength() * sizeof(float), hipMemcpyDeviceToDevice);
}
Mat::Mat(int height, int width, int nchannels){
	cols = width;
	rows = height;
	channels = nchannels;
	hostData = NULL;
	devData = NULL;
	mallocHost();
	mallocDevice();
	zeros();
}
Mat::~Mat(){
	if(NULL != hostData)
		MemoryMonitor::instance()->freeCpuMemory(hostData);
	if(NULL != devData)
		MemoryMonitor::instance()->freeGpuMemory(devData);
}

Mat& Mat::operator=(const Mat &m){
	cols = m.cols;
	rows = m.rows;
	channels = m.channels;
	hostData = NULL;
	devData = NULL;
	mallocHost();
	mallocDevice();
	memcpy(hostData, m.hostData, getLength() * sizeof(float));
	hipMemcpy(devData, m.devData, getLength() * sizeof(float), hipMemcpyDeviceToDevice);
    return *this;
}

void Mat::zeros(){
	setAll(0.0);
}

void Mat::ones(){
	setAll(1.0);
}

void Mat::randn(){
	if(NULL == hostData) mallocHost();
	if(NULL == devData) mallocDevice();
	hiprandGenerator_t gen;
	// Create pseudo-random number generator
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	// Set seed
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
	// Generate n floats on device
	hiprandGenerateUniform(gen, devData, getLength());
	// Cleanup generator
	hiprandDestroyGenerator(gen);
	deviceToHost();
}

void Mat::set(int pos_y, int pos_x, int pos_channel, float val){
	if(NULL == hostData || NULL == devData) {zeros();}
	if(pos_x >= cols || pos_y >= rows || pos_channel >= channels){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	hostData[IDX2C(pos_y, pos_x, cols) + pos_channel * (rows * cols)] = val;
	hostToDevice();
}

void Mat::setAll(float val){
	if(NULL == hostData) {mallocHost();}
	if(NULL == devData) {mallocDevice();}
	int len = getLength();
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	cu_setAll<<<num_blocks, block_size>>>(devData, val, len);
	deviceToHost();
}

void Mat::setAll(const vector3f &v){
	if(channels != 3){
		std::cout<<"this is not a 3 channel mat..."<<std::endl;
		exit(0);
	}
	if(NULL == hostData) {mallocHost();}
	if(NULL == devData) {mallocDevice();}
	int len = rows * cols;
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	for(int i = 0; i < channels; ++i){
		cu_setAll<<<num_blocks, block_size>>>(devData + i * len, v.get(i), len);
	}
	deviceToHost();
}

float Mat::get(int pos_y, int pos_x, int pos_channel) const{
	if(NULL == hostData || NULL == devData||
	   pos_x >= cols || pos_y >= rows || pos_channel >= channels){
		std::cout<<"invalid position..."<<std::endl;
		exit(0);
	}
	return hostData[IDX2C(pos_y, pos_x, cols) + pos_channel * (rows * cols)];
}

int Mat::getLength() const{
	return rows * cols * channels;
}

void Mat::deviceToHost(){
	if(NULL == hostData) mallocHost();
	if(NULL == devData) mallocDevice();
	// Copy device memory to host
	hipMemcpy(hostData, devData, getLength() * sizeof(float), hipMemcpyDeviceToHost);
}

void Mat::hostToDevice(){
	if(NULL == hostData) mallocHost();
	if(NULL == devData) mallocDevice();
	// Copy host memory to device
	hipMemcpy(devData, hostData, getLength() * sizeof(float), hipMemcpyHostToDevice);
}

void Mat::copyTo(Mat &m){
	m.rows = rows;
	m.cols = cols;
	m.channels = channels;
	m.hostData = NULL;
	m.devData = NULL;
	m.mallocHost();
	m.mallocDevice();
	memcpy(m.hostData, hostData, getLength() * sizeof(float));
	hipMemcpy(m.devData, devData, getLength() * sizeof(float), hipMemcpyDeviceToDevice);
}

// only changes devData (on GPU)
Mat Mat::operator+(const Mat &m){
	if(NULL == hostData || NULL == devData ||
	   NULL == m.hostData || NULL == m.devData||
	   getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	Mat tmpmat(m);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	hipblasSetVector (n, sizeof (float), hostData, 1, devData, 1); // cp x- >d_x
	hipblasSetVector (n, sizeof (float), tmpmat.hostData, 1, tmpmat.devData, 1); // cp y- >d_y
	float alpha = 1.0;
	// multiply the vector d_x by the scalar alpha and add to d_y
	hipblasSaxpy(handle, n, &alpha, devData, 1, tmpmat.devData, 1);
	hipblasGetVector (n, sizeof (float), tmpmat.devData, 1, tmpmat.hostData, 1); // cp d_y - >y
	hipblasDestroy ( handle ); // destroy CUBLAS context
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator+(float val){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	int len = getLength();
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	cu_plus<<<num_blocks, block_size>>>(tmpmat.devData, val, len);
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator+(const vector3f &v){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	if(channels != 3){
		std::cout<<"this is not a 3 channel mat..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	int len = rows * cols;
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	for(int i = 0; i < channels; ++i){
		float tmp = v.get(i);
		cu_plus<<<num_blocks, block_size>>>(tmpmat.devData + i * len, tmp, len);
	}
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator-(const Mat &m){

	if(NULL == hostData || NULL == devData ||
	   NULL == m.hostData || NULL == m.devData||
	   getLength() != m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	Mat tmpmat;
	copyTo(tmpmat);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	hipblasSetVector (n, sizeof (float), m.hostData, 1, m.devData, 1); // cp x- >d_x
	hipblasSetVector (n, sizeof (float), tmpmat.hostData, 1, tmpmat.devData, 1); // cp y- >d_y
	float alpha = -1.0;
	// multiply the vector d_x by the scalar alpha and add to d_y
	hipblasSaxpy(handle, n, &alpha, m.devData, 1, tmpmat.devData, 1);
	hipblasGetVector (n, sizeof (float) ,tmpmat.devData, 1, tmpmat.hostData, 1); // cp d_y - >y
	hipblasDestroy ( handle ); // destroy CUBLAS context
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator-(float val){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	int tmp = getLength();
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (tmp / block_size) + ((tmp % block_size) ? 1 : 0);
	cu_minus<<<num_blocks, block_size>>>(tmpmat.devData, val, tmp);
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator-(const vector3f &v){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	if(channels != 3){
		std::cout<<"this is not a 3 channel mat..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	int len = rows * cols;
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (len / block_size) + ((len % block_size) ? 1 : 0);
	for(int i = 0; i < channels; ++i){
		cu_minus<<<num_blocks, block_size>>>(tmpmat.devData + i * len, v.get(i), len);
	}
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator*(const Mat &m){
	if(NULL == hostData || NULL == devData ||
	   NULL == m.hostData || NULL == m.devData||
	   cols != m.rows || channels != m.channels){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	Mat tmpmat(rows, m.cols, channels);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	float alpha = 1.0;
	float beta = 1.0;
	for(int i = 0; i < channels; ++i){
		hipblasSetMatrix (rows, cols, sizeof(float), hostData + i * (rows * cols), rows, devData + i * (rows * cols), rows); // cp x- >d_x
		hipblasSetMatrix (m.rows, m.cols, sizeof(float), m.hostData + i * (m.rows * m.cols), m.rows, m.devData + i * (m.rows * m.cols), m.rows); // cp y- >d_y
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, m.cols, cols, &alpha, devData + i * (rows * cols), rows, m.devData + i * (m.rows * m.cols), cols, &beta, tmpmat.devData + i * (tmpmat.rows * tmpmat.cols), rows);
		hipblasGetMatrix (rows, m.cols, sizeof(float), tmpmat.devData + i * (tmpmat.rows * tmpmat.cols), rows, tmpmat.hostData + i * (tmpmat.rows * tmpmat.cols), rows);
	}
	hipblasDestroy (handle); // destroy CUBLAS context
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::operator*(float val){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	Mat tmpmat;
	copyTo(tmpmat);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	hipblasSscal(handle, n, &val, tmpmat.devData, 1);
	tmpmat.deviceToHost();
    hipblasDestroy(handle);
	return tmpmat;
}

Mat Mat::operator*(const vector3f &v){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	if(channels != 3){
		std::cout<<"this is not a 3 channel mat..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	for(int i = 0; i < channels; ++i){
		float tmp = v.get(i);
		hipblasSscal(handle, rows * cols, &tmp, tmpmat.devData + i * rows * cols, 1);
	}
	tmpmat.deviceToHost();
    hipblasDestroy(handle);
	return tmpmat;
}

Mat Mat::mul(const Mat &m){
	if(NULL == hostData || NULL == devData ||
	   NULL == m.hostData || NULL == m.devData||
	   getLength()!= m.getLength()){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	int tmp = getLength();
	const size_t block_size = threadsPerBlock;
	const size_t num_blocks = (tmp / block_size) + ((tmp % block_size) ? 1 : 0);
	cu_elementWiseMultiply<<<num_blocks, block_size>>>(devData, m.devData, tmpmat.devData, tmp);
	tmpmat.deviceToHost();
	return tmpmat;
}

Mat Mat::mul(float val){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	int n = getLength();
	Mat tmpmat;
	copyTo(tmpmat);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	hipblasSscal(handle, n, &val, tmpmat.devData, 1);
	tmpmat.deviceToHost();
    hipblasDestroy(handle);
	return tmpmat;
}

Mat Mat::mul(const vector3f &v){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	if(channels != 3){
		std::cout<<"this is not a 3 channel mat..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	for(int i = 0; i < channels; ++i){
		float tmp = v.get(i);
		hipblasSscal(handle, rows * cols, &tmp, tmpmat.devData + i * rows * cols, 1);
	}
	tmpmat.deviceToHost();
    hipblasDestroy(handle);
	return tmpmat;
}

Mat Mat::t(){
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid vectors..."<<std::endl;
		exit(0);
	}
	Mat tmpmat;
	copyTo(tmpmat);
	//tmpmat.zeros();
    float const alpha(1.0);
    float const beta(0.0);
	hipblasHandle_t handle; // CUBLAS context
	hipblasCreate (&handle); // initialize CUBLAS context
	for(int i = 0; i < channels; ++i){
		hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, rows, &alpha, devData + i * (rows * cols), rows, &beta, devData + i * (rows * cols), cols, tmpmat.devData + i * (rows * cols), cols);
	}
	int tmp = tmpmat.rows;
	tmpmat.rows = tmpmat.cols;
	tmpmat.cols = tmp;
	hipblasDestroy(handle);
	tmpmat.deviceToHost();
	return tmpmat;
}

// memory
void Mat::mallocHost(){
	if(NULL == hostData){
		// malloc host data
		hostData = (float*)MemoryMonitor::instance()->cpuMalloc(cols * rows * channels * sizeof(float));
		if(NULL == hostData) {
			std::cout<<"host memory allocation failed..."<<std::endl;
			exit(0);
		}
		memset(hostData, 0, cols * rows * channels * sizeof(float));
	}
}

void Mat::mallocDevice(){
	if(NULL == devData){
		hipError_t cudaStat;
		// malloc device data
		cudaStat = MemoryMonitor::instance()->gpuMalloc((void**)&devData, cols * rows * channels * sizeof(float));
		if(cudaStat != hipSuccess) {
			std::cout<<"device memory allocation failed..."<<std::endl;
			exit(0);
		}
		cudaStat = hipMemset(devData, 0, sizeof(float) * cols * rows * channels);
		if(cudaStat != hipSuccess) {
			std::cout<<"device memory hipMemset failed..."<<std::endl;
			exit(0);
		}
	}
}

void Mat::printHost(const std::string &str){
	std::cout<<str<<std::endl;
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid matrix..."<<std::endl;
		exit(0);
	}
	Mat show = t();
	int counter = 0;
	std::cout<<"Matrix with "<<channels<<" channels, "<<rows<<" rows, "<<cols<<"columns."<<std::endl;
	for(int i = 0; i < channels; ++i){
		std::cout<<"Channel "<<i<<" : "<<std::endl;
		for(int j = 0; j < rows; ++j){
			for(int k = 0; k < cols; ++k){
				std::cout<<show.hostData[counter]<<" ";
				++ counter;
			}
			std::cout<<std::endl;
		}
	}
}

void Mat::printDevice(const std::string &str){
	std::cout<<str<<std::endl;
	if(NULL == hostData || NULL == devData){
		std::cout<<"invalid matrix..."<<std::endl;
		exit(0);
	}
	Mat show = t();
	float *host_data = 0;
	host_data = (float*)MemoryMonitor::instance()->cpuMalloc(show.cols * show.rows * show.channels * sizeof(float));
	hipMemcpy(host_data, show.devData, show.cols * show.rows * show.channels * sizeof(float), hipMemcpyDeviceToHost);
	int counter = 0;
	std::cout<<"Matrix with "<<channels<<" channels, "<<rows<<" rows, "<<cols<<"columns."<<std::endl;
	for(int i = 0; i < channels; ++i){
		std::cout<<"Channel "<<i<<" : "<<std::endl;
		for(int j = 0; j < rows; ++j){
			for(int k = 0; k < cols; ++k){
				std::cout<<host_data[counter]<<" ";
				++ counter;
			}
			std::cout<<std::endl;
		}
	}
	if(NULL != host_data)
		MemoryMonitor::instance()->freeCpuMemory(host_data);
}
